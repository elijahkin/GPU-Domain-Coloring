#include "hip/hip_runtime.h"
#include "utils.cu"

__global__ void domain_color_kernel(Function f, int N, int width, double min_re,
                                    double max_im, double step_size,
                                    uint8_t *rgb) {
  int offset = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (int n = offset; n < N; n += stride) {
    // Get this pixel's row and col indices
    int row = n / width;
    int col = n % width;

    // Convert indices to complex number
    Complex z(min_re + col * step_size, max_im - row * step_size);

    // Apply the function
    Complex w = f(z);

    // Convert result to rgb and write to memory
    complex_to_rgb(w, rgb, n);
  }
}

void domain_color(Function f, Complex center, double apothem_real, int width,
                  int height) {
  // Allocate memory for storing pixels
  uint8_t *rgb;
  hipMallocManaged(&rgb, width * height * 3 * sizeof(uint8_t));

  // Calculate derived constants
  int N = width * height;
  double min_real = center.real() - apothem_real;
  double max_imag = center.imag() + (apothem_real * height) / width;
  double step_size = 2.0 * apothem_real / (width - 1);

  // These blocks and thread numbers were chosen for my RTX 3060
  domain_color_kernel<<<28, 128>>>(f, N, width, min_real, max_imag, step_size,
                                   rgb);

  char filename[100];
  sprintf(filename, "renders/domain_color_%s.png", "happy");
  save_png(rgb, width, height, filename);
}

int main() {
  auto identity = [] __device__(Complex z) -> Complex {
    return z;
  };
  domain_color(identity, 0, 2, 1024, 1024);
  return 0;
}