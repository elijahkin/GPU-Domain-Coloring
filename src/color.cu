
#include <hip/hip_runtime.h>
#include <cstdint>

struct Color {
  uint8_t r;
  uint8_t g;
  uint8_t b;
};

__device__ Color hsl_to_rgb(double h, double s, double l) {
  double chroma = (1 - abs(2 * l - 1)) * s;
  double x = chroma * (1 - abs(fmod(6 * h, 2.0) - 1));
  double m = l - (chroma / 2.0);

  uint8_t sextant = int(h * 6);
  double r, g, b;
  switch (sextant) {
  case 0:
    r = chroma;
    g = x;
    b = 0;
    break;
  case 1:
    r = x;
    g = chroma;
    b = 0;
    break;
  case 2:
    r = 0;
    g = chroma;
    b = x;
    break;
  case 3:
    r = 0;
    g = x;
    b = chroma;
    break;
  case 4:
    r = x;
    g = 0;
    b = chroma;
    break;
  case 5:
    r = chroma;
    g = 0;
    b = x;
    break;
  default:
    r = g = b = 0;
    break;
  }
  return {(uint8_t)((r + m) * 255), (uint8_t)((g + m) * 255),
          (uint8_t)((b + m) * 255)};
}

__device__ uint8_t lerp(uint8_t a, uint8_t b, double t) {
  return static_cast<uint8_t>(a + t * (b - a));
}

__device__ Color color_lerp(Color x, Color y, double t) {
  return {lerp(x.r, y.r, t), lerp(x.g, y.g, t), lerp(x.b, y.b, t)};
}
