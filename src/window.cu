#include "hip/hip_runtime.h"
#include "complex_plot.cu"
#include <GL/glut.h>

// Global variables
float center_re = 0.0;
float center_im = 0.0;
float apothem_re = 1.0;

uint8_t *rgb;
Image render;
GLuint textureID;

int screenWidth, screenHeight;
int lastMouseX, lastMouseY;
bool mouseLeftDown = false;
bool displayInfo = false;

void mouse(int button, int state, int x, int y) {
  if (button == GLUT_LEFT_BUTTON) {
    if (state == GLUT_DOWN) {
      mouseLeftDown = true;
      lastMouseX = x;
      lastMouseY = y;
    } else if (state == GLUT_UP) {
      mouseLeftDown = false;
    }
  } else if (button == 3) { // Scroll up
    apothem_re *= 1.1;
  } else if (button == 4) { // Scroll down
    apothem_re *= 0.9;
  }
  glutPostRedisplay();
}

void motion(int x, int y) {
  if (mouseLeftDown) {
    int deltaX = x - lastMouseX;
    int deltaY = y - lastMouseY;

    center_re -= 2 * deltaX * apothem_re / screenWidth;
    center_im -= 2 * deltaY * apothem_re / screenWidth;

    // Update the last mouse position to the current position
    lastMouseX = x;
    lastMouseY = y;

    glutPostRedisplay();
  }
}

void keyboard(unsigned char key, int x, int y) {
  switch (key) {
  case 27: // Escape key
    hipFree(rgb);
    exit(0);
    break;
  case 32: // Space
    center_re = 0.0;
    center_im = 0.0;
    apothem_re = 1.0;
    glutPostRedisplay();
    break;
  case 96: // `
    displayInfo = !displayInfo;
    glutPostRedisplay();
    break;
  }
}

void drawString(float x, float y, float z, std::string &text) {
  // Save the current attributes (including color)
  glPushAttrib(GL_CURRENT_BIT);

  // Set text color to white
  glColor3f(0.0f, 0.0f, 0.0f);

  // Set initial raster position
  glRasterPos3f(x, y, z);

  for (char c : text) {
    if (c == '\n') {
      // Move to the next line
      y -= 0.05f;             // Adjust the line spacing as needed
      glRasterPos3f(x, y, z); // Set new raster position for the next line
    } else {
      // Render the character
      glutBitmapCharacter(GLUT_BITMAP_9_BY_15, c);
    }
  }
  glPopAttrib();
}

// Create info string to display if displayInfo is true
std::string getInfoString() {
  return "Mouse X: " + std::to_string(lastMouseX) +
         "\nMouse Y: " + std::to_string(lastMouseY) +
         "\nCenter X: " + std::to_string(center_re) +
         "\nCenter Y: " + std::to_string(center_im) +
         "\nApothem: " + std::to_string(apothem_re);
}

void display() {
    // Use CUDA kernel to render function
  float min_re = center_re - apothem_re;
  float max_re = center_re + apothem_re;
  float apothem_im = screenHeight * apothem_re / screenWidth;
  float max_im = center_im + apothem_im;
  float min_im = center_im - apothem_im;
  float step_size = 2 * apothem_re / (screenWidth - 1);

  domain_color_kernel<<<28, 128>>>(
      [] __device__(Complex z) { return pow(z, 3) - 1; }, render, min_re,
      max_im, step_size);

  // Image pattern = read_ppm("patterns/cannon.ppm");
  // conformal_map_kernel<<<28, 128>>>(
  //     [] __device__(Complex z) { return pow(z, -2); }, render, min_re, max_re,
  //     min_im, max_im, step_size, pattern);

  // escape_time_kernel<<<28, 128>>>(
  //     [] __device__(Complex z, Complex c) { return pow(z, 7) + c; }, render,
  //     min_re, max_im, step_size, 10);
  hipDeviceSynchronize();

  // Update OpenGL texture with CUDA output
  glBindTexture(GL_TEXTURE_2D, textureID);
  glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, render.width, render.height, GL_RGB,
                  GL_UNSIGNED_BYTE, render.rgb);

  // Clear the color buffer and set up for rendering
  glClear(GL_COLOR_BUFFER_BIT);
  glEnable(GL_TEXTURE_2D);

  // Draw textured rectangle
  glBegin(GL_QUADS);
  glTexCoord2f(0.0f, 0.0f);
  glVertex2f(-1.0f, -1.0f);
  glTexCoord2f(1.0f, 0.0f);
  glVertex2f(1.0f, -1.0f);
  glTexCoord2f(1.0f, 1.0f);
  glVertex2f(1.0f, 1.0f);
  glTexCoord2f(0.0f, 1.0f);
  glVertex2f(-1.0f, 1.0f);
  glEnd();

  // Disable texture mapping and swap buffers
  glDisable(GL_TEXTURE_2D);

  // Draw info string for debugging
  if (displayInfo) {
    std::string info = getInfoString();
    drawString(-0.98, 0.95, 0.0, info);
  }

  glutSwapBuffers();
}

int main(int argc, char **argv) {
  // Initialize GLUT
  glutInit(&argc, argv);

  // Set display mode
  glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB);

  // Get screen size
  screenWidth = glutGet(GLUT_SCREEN_WIDTH);
  screenHeight = glutGet(GLUT_SCREEN_HEIGHT);

  // Allocate CUDA memory
  hipMallocManaged(&rgb, screenWidth * screenHeight * 3 * sizeof(uint8_t));
  render = {screenWidth, screenHeight, screenWidth * screenHeight, rgb};

  // Set up window size and name
  glutInitWindowSize(screenWidth, screenHeight);
  glutCreateWindow("Functiongram");

  // Set fullscreen
  glutFullScreen();

  // Set up OpenGL for texture mapping
  glEnable(GL_TEXTURE_2D);

  // Set texture parameters
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP);

  // Specify texture image data
  glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB, screenWidth, screenHeight, 0, GL_RGB,
               GL_UNSIGNED_BYTE, rgb);

  // Set up the callback functions
  glutDisplayFunc(display);
  glutKeyboardFunc(keyboard);
  glutMouseFunc(mouse);
  glutMotionFunc(motion);

  // Enter the GLUT event processing loop
  glutMainLoop();

  return 0;
}
